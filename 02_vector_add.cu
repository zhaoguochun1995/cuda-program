#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include <iostream>

#define CALL_CUDA(expr) \
    do {    \
        hipError_t code = expr; \
        if (code != hipSuccess)  { \
        } \
    } while(0);

template <typename T>
void vectorAddCPU(const T* a, const T* b, T* out, int N) {
    for (int i = 0; i < N; ++i) {
        out[i] = a[i] + b[i];
    }
} 


template <typename T>
__global__ void vectorAddCUDA(const T* a, const T* b, T* out, int N) {
    for (int i = 0; i < N; ++i) {
        out[i] = a[i] + b[i];
    }
} 

int main(int argc, char** argv) {
    using T = float;
    const int N = 1024;
    std::vector<T> a_h(N);
    std::vector<T> b_h(N);
    std::vector<T> c_h(N);

    for (size_t i = 0; i < N; ++i) {
        a_h[i] = std::sin(i);
        b_h[i] = std::cos(i) * 2;
        c_h[0] = 0;
    }

    T* a_d = nullptr;
    T* b_d = nullptr;
    T* c_d = nullptr;
    CALL_CUDA(hipMalloc(&a_d, sizeof(T) * N));
    CALL_CUDA(hipMalloc(&b_d, sizeof(T) * N));
    CALL_CUDA(hipMalloc(&c_d, sizeof(T) * N));

    CALL_CUDA(hipMemcpy(a_d, a_h.data(), sizeof(T) * N, hipMemcpyHostToDevice));
    CALL_CUDA(hipMemcpy(b_d, b_h.data(), sizeof(T) * N, hipMemcpyHostToDevice));

    vectorAddCPU(a_h.data(), b_h.data(), c_h.data(), N);

    //Dim blockDim(256, 1, 1);
    int block_size = 256;
    int grid_size = (N - block_size + 1) / block_size;
    vectorAddCUDA<<<grid_size, block_size>>>(a_d, b_d, c_d, N);

    CALL_CUDA(hipDeviceSynchronize());

    std::vector<T> c_d_ref(N);
    CALL_CUDA(hipMemcpy(c_d_ref.data(), c_d, sizeof(T) * N, hipMemcpyDeviceToHost));
    
    T max_diff = 0;
    for (size_t i = 0; i < N; i++) {
        T diff = std::abs(c_h[i] - c_d_ref[i]);
        std::cout << c_h[i] << "\t" << c_d_ref[i] << "\t" << diff << std::endl;
        max_diff = std::max(max_diff, diff);
    }
    std::cout << "cpu and gpu max diff:" << max_diff << std::endl;


    CALL_CUDA(hipFree(a_d));
    CALL_CUDA(hipFree(b_d));
    CALL_CUDA(hipFree(c_d));
    CALL_CUDA(hipDeviceReset());
}